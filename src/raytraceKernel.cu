#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
#include <hip/hip_vector_types.h>
#else
#include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		exit(EXIT_FAILURE); 
	}
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
	int index = x + (y * resolution.x);

	thrust::default_random_engine rng(hash(index*time));
	thrust::uniform_real_distribution<float> u01(0,1);

	return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Function that traverses scene searching for collisions. Traces ray to first impact. Returns index of first geometry hit or -1 if no collision
__host__ __device__ int firstIntersect(staticGeom* geoms, int numberOfGeoms, ray r, glm::vec3& intersectionPoint, glm::vec3& normal, float& distance)
{
	//Index of the first hit geometry
	int firstGeomInd = -1;
	distance = -1;
	//Best intersection points stored in output params as minimums encountered. Limits temporary variables..

	//for each geometry object
	//TODO create better scene graph to improve collision detection for more complicated scenes. (Octtree)
	for(int i = 0; i < numberOfGeoms; ++i)
	{
		//Temporary return variables
		glm::vec3 intersectionPointTemp;
		glm::vec3 normalTemp;

		//Test for collision
		float dist = geomIntersectionTest(geoms[i], r, intersectionPointTemp, normalTemp);
		if(dist > RAY_BIAS_AMOUNT)//Exclude very very near hits. This should help fix floating point errors.
		{
			//Impact detected
			if(distance < 0 || dist < distance)
			{
				//First hit or closer hit
				distance = dist;
				firstGeomInd = i;
				intersectionPoint = intersectionPointTemp;
				normal = normalTemp;
			}
		}
	}

	return firstGeomInd;
}

__host__ __device__ glm::vec3 reflect(glm::vec3 incident, glm::vec3 normal)
{
	return incident-glm::dot(2.0f*normal, incident) * normal;
}

///Compute the scalar contribution from specular highlights
__host__ __device__ float calculateSpecularScalar(ray viewRay, ray lightDirection, glm::vec3 normal, float specularExponent)
{
	float dot = glm::dot(reflect(-lightDirection.direction, normal), -viewRay.direction);
	if(dot <= 0.0)
		return 0.0;
	else
		return glm::pow(dot, specularExponent);
}

///Compute the scalar contribution from diffuse lighting
__host__ __device__ float calculateDiffuseScalar(glm::vec3 normal, ray lightDirection)
{
	return MAX(0,glm::dot(normal, lightDirection.direction));
}

__host__ __device__ glm::vec3 traceShadowRay(ray feeler, renderOptions rconfig, staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials, int targetLight)
{
	glm::vec3 lightHitPoint;
	float lightDistance;
	glm::vec3 normal;
	if(firstIntersect(geoms, numberOfGeoms, feeler, lightHitPoint, normal, lightDistance) == targetLight)
		//Return color of light
			return mats[geoms[targetLight].materialid].color*mats[geoms[targetLight].materialid].emittance;
	else
		//Shadow
		return glm::vec3(0,0,0);
}

__host__ __device__ glm::vec3 computeShadowedIntensity(ray primeRay, glm::vec3 intersectionPoint, int hitGeomIndex, glm::vec3 normal, renderOptions rconfig, 
													   float seed,	staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials, int lightIndex, ray& lightDirection, bool softShadowRegion)
{
	//TODO: implement soft shadows
	material lightMat =  mats[geoms[lightIndex].materialid];
	if(softShadowRegion && rconfig.softShadows){
		glm::vec3 color = glm::vec3(0,0,0);
		glm::vec3 shadowContrib;
		if(rconfig.parallelShadows)
		{

		}else{

			glm::vec3 avgDirection = glm::vec3(0,0,0);
			lightDirection.origin = intersectionPoint;
			for(int i = 0; i < rconfig.numShadowRays; ++i)
			{

				glm::vec3 randomEndpoint;
				if(geoms[lightIndex].type == CUBE){
					randomEndpoint = getRandomPointOnCube(geoms[lightIndex], seed*(i+1));
				}else{
					randomEndpoint = getRandomPointOnSphere(geoms[lightIndex], seed*(i+1));
				}

				lightDirection.direction = glm::normalize(randomEndpoint - intersectionPoint);

				shadowContrib = traceShadowRay(lightDirection, rconfig, geoms, numberOfGeoms, mats, numberOfMaterials, lightIndex);
				if(shadowContrib.x > 0 || shadowContrib.y > 0 || shadowContrib.z > 0)
				{
					color+= shadowContrib/(float)rconfig.numShadowRays;
					avgDirection += lightDirection.direction;

				}
			}
		}
		lightDirection.direction = glm::normalize(lightDirection.direction);
		return color;
	}
	else{
		//For now just return the center of the light source. Treat as a point source

		lightDirection.origin = intersectionPoint;
		lightDirection.direction = glm::normalize(geoms[lightIndex].translation - intersectionPoint);

		return traceShadowRay(lightDirection, rconfig, geoms, numberOfGeoms, mats, numberOfMaterials, lightIndex);

	}
}

//Computes the illumination contributions from each light source at this point.
//Incorporates ambient, specular, and diffuse reflection as well as shadows.
//Returns the summed light intensity in rgb components. Perfect reflection and refraction effects are not included.
__host__ __device__ glm::vec3 calculatePhongIllumination(ray primeRay, glm::vec3 intersectionPoint, int hitGeomIndex, glm::vec3 normal, renderOptions rconfig, 
														 float seed,	staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials, bool softShadowRegion)
{
	//Initialize to ambient component.
	glm::vec3 totalL = rconfig.ka*rconfig.ambientLight*mats[geoms[hitGeomIndex].materialid].color;

	//for each material, if it's a light source add its acumulated 
	//TODO: precompute which objects are light sources
	for(int i = 0; i < numberOfGeoms; ++i){
		if(mats[geoms[i].materialid].emittance > 0)
		{
			if(i == hitGeomIndex)
			{
				//we hit a light, add in its own component
				totalL += mats[geoms[i].materialid].emittance*mats[geoms[i].materialid].color;
			}else{
				//Light source, compute contribution
				ray lightDirection;//An output variable that returns the direction to the center of the effective light source
				glm::vec3 lightIntensity = computeShadowedIntensity(primeRay, intersectionPoint, hitGeomIndex, normal, rconfig, 
					seed,	geoms, numberOfGeoms, mats, numberOfMaterials, i, lightDirection, softShadowRegion);
				if(rconfig.mode == SHADOW_DEBUG)
				{
					totalL += lightIntensity;
				}else{
					if(lightIntensity.x > 0 || lightIntensity.y > 0 || lightIntensity.z > 0){
						//Compute diffuse contribution
						if(rconfig.kd > 0)
						{

							//kd is a global tuning parameter that allows control of each lighting element.
							totalL += rconfig.kd*(lightIntensity*mats[geoms[hitGeomIndex].materialid].color)
								*calculateDiffuseScalar(normal, lightDirection);
						}

						//Compute Specular contribution (only if non-zero specular exponent, otherwise everything will go bright)
						if(rconfig.ks > 0 && mats[geoms[hitGeomIndex].materialid].specularExponent > 0)
						{
							totalL += rconfig.ks*(lightIntensity*mats[geoms[hitGeomIndex].materialid].specularColor)
								*calculateSpecularScalar(primeRay, lightDirection, normal, mats[geoms[hitGeomIndex].materialid].specularExponent);
						}
					}
				}
			}
		}
	}

	return totalL;
}


//TODO: verify raycastFromCameraKernel FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, float x, float y, glm::vec3 eye, 
												glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	ray r;
	r.origin = eye;
	glm::vec3 right = glm::cross(view, up);

	//float d = 1.0f; use a viewing plane of 1 distance 
	glm::vec3 pixel_location = /* d* */(view + (2*x/resolution.x-1)*right*glm::tan(glm::radians(fov.x)) 
		- (2*y/resolution.y-1)*up*glm::tan(glm::radians(fov.y)));

	r.direction = glm::normalize(pixel_location);

	return r;
}

__host__ __device__ glm::vec3 traceRay(ray primeRay, float seed, renderOptions rconfig, 
									   staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials, bool softShadowRegion)
{
	glm::vec3 color;
	//First we must have a primary ray
	//Calculate impact of primary ray
	float dist;
	glm::vec3 intersectionPoint;
	glm::vec3 normal;
	int ind = firstIntersect(geoms, numberOfGeoms, primeRay, intersectionPoint, normal, dist);

	if(ind >= 0)
	{
		//We have something to draw. 
		switch(rconfig.mode)
		{
		case NORMAL_DEBUG:
			//Debug render. Display normals of very first impacted surface.
			color = glm::abs(normal);
			break;
		case DISTANCE_DEBUG:
			color = glm::vec3(1,1,1)*(1-dist/rconfig.distanceShadeRange);
			break;
		case RAYTRACE:
		case ALIASING_DEBUG:
		case SHADOW_DEBUG:
			//TODO Implement actual raytracer here
			color = calculatePhongIllumination(primeRay, intersectionPoint, ind, normal, rconfig, seed, 
				geoms, numberOfGeoms, mats, numberOfMaterials, softShadowRegion);
			break;
		}

	}

	return color;
}



__host__ __device__ int estimateNumSamples(int x, int y, glm::vec2 resolution, glm::vec3* colors, renderOptions rconfig)
{
	//TODO implement more flexible options

	//Compute RMSD in local window 3x3
	int n = 0;
	glm::vec3 accumulator = glm::vec3(0,0,0);
	for(int yi = MAX(0,y - 1); yi <= MIN(y + 1, resolution.y-1); ++yi)
	{
		for(int xi = MAX(0,x - 1); xi <= MIN(x + 1, resolution.x-1); ++xi)
		{
			++n;
			int index = xi + (yi * resolution.x);
			accumulator += colors[index];
		}
	}

	glm::vec3 mean = accumulator/(float)n;
	accumulator = glm::vec3(0,0,0);


	for(int yi = MAX(0,y - 1); yi <= MIN(y + 1, resolution.y-1); ++yi)
	{
		for(int xi = MAX(0,x - 1); xi <= MIN(x + 1, resolution.x-1); ++xi)
		{
			int index = xi + (yi * resolution.x);
			accumulator += (colors[index]-mean)*(colors[index]-mean);
		}
	}

	glm::vec3 RMSD = glm::sqrt(accumulator/(float)n);

	if(RMSD.x > rconfig.aargbThresholds.x || RMSD.y > rconfig.aargbThresholds.y || RMSD.z > rconfig.aargbThresholds.z)
	{
		return rconfig.maxSamplesPerPixel;
	}
	else
	{
		return rconfig.minSamplesPerPixel;
	}

}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	if(x<resolution.x && y<resolution.y){
		image[index] = glm::vec3(0,0,0);
	}
}


//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<resolution.x && y<resolution.y){

		glm::vec3 color;
		color.x = image[index].x*255.0;
		color.y = image[index].y*255.0;
		color.z = image[index].z*255.0;

		if(color.x>255){
			color.x = 255;
		}

		if(color.y>255){
			color.y = 255;
		}

		if(color.z>255){
			color.z = 255;
		}

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 0;
		PBOpos[index].x = color.x;
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}

__global__ void estimateSamples(glm::vec2 resolution, renderOptions rconfig, glm::vec3* colors, int* numSamples)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	numSamples[index] = estimateNumSamples(x,y,resolution,colors, rconfig);
}


__global__ void drawOverSamples(glm::vec2 resolution, glm::vec3* colors, int* numSamples, renderOptions rconfig)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(numSamples[index] > rconfig.minSamplesPerPixel){
		colors[index] = glm::vec3(0,1,0);
	}
}


//TODO: IMPLEMENT raytraceRay Kernel FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, renderOptions rconfig, glm::vec3* colors, int* numSamples,
							staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	float seed = (time+index);
	if((x<resolution.x && y<resolution.y)){  
		//Valid pixel, away we go!

		bool softShadowRegion = numSamples[index] > rconfig.minSamplesPerPixel;
		if(rconfig.antialiasing){
			thrust::default_random_engine rng(seed);
			thrust::uniform_real_distribution<float> u0505(-0.5,0.5);

			for(int i = 0; i < numSamples[index]; ++i)
			{

				ray primeRay = raycastFromCameraKernel(resolution, seed, x+u0505(rng), y+u0505(rng), cam.position, cam.view, cam.up, cam.fov);
				colors[index] += traceRay(primeRay, seed, rconfig, geoms, numberOfGeoms, mats, numberOfMaterials, softShadowRegion)/((float)numSamples[index]);

			}

		}else{
			//simply cast a single ray
			ray primeRay = raycastFromCameraKernel(resolution, seed, x, y, cam.position, cam.view, cam.up, cam.fov);
			colors[index] = traceRay(primeRay, seed, rconfig, geoms, numberOfGeoms, mats, numberOfMaterials, softShadowRegion);
		}
	}
}

//TODO: FINISH Kernel Wrapper FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, renderOptions* renderOpts, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){

	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), 
		(int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	//send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

	//Array to hold samples per pixel (for adaptive anti-aliasing)
	int* cudasamples = NULL;
	hipMalloc((void**)&cudasamples, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	//Don't need to copy anything, it just stays on GPU

	//package geometry and materials and sent to GPU
	staticGeom* geomList = new staticGeom[numberOfGeoms];
	for(int i=0; i<numberOfGeoms; i++){
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[i] = newStaticGeom;
	}

	//Debug code
	//getRandomPointOnSphere(geomList[7], (float)iterations);
	staticGeom* cudageoms = NULL;
	hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
	hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	material* cudamats = NULL;
	hipMalloc((void**)&cudamats, numberOfMaterials*sizeof(material));
	hipMemcpy( cudamats, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);


	//package camera
	cameraData cam;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->positions[frame];
	cam.view = renderCam->views[frame];
	cam.up = renderCam->ups[frame];
	cam.fov = renderCam->fov;


	ray r;
	r.origin = glm::vec3(0,0,10);
	r.direction = glm::vec3(0,0,-1);
	glm::vec3 intersectionPoint;
	glm::vec3 normal;
	float result = boxIntersectionTest(geomList[0], r, intersectionPoint, normal);

	//Always run just to be safe.
	estimateSamples<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, *renderOpts, cudaimage, cudasamples);

	clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage);

	//kernel launches
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, *renderOpts, cudaimage, cudasamples, cudageoms, numberOfGeoms, cudamats, numberOfMaterials);

	//retrieve image from GPU
	hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

	if(renderOpts->mode == ALIASING_DEBUG)
	{
		drawOverSamples<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, cudasamples, *renderOpts);
	}

	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

	//free up stuff, or else we'll leak memory like a madman
	hipFree( cudaimage );
	hipFree( cudasamples);
	hipFree( cudamats  );
	hipFree( cudageoms );
	delete geomList;

	// make certain the kernel has completed
	hipDeviceSynchronize();

	checkCUDAError("Kernel failed!");
}
